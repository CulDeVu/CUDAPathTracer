#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <chrono>
#include <hip/hip_runtime.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <math.h>

#include "modelLoader.h"

#include "BVH.h"
#include "lens.h"
#include "color.h"
#include "sphere.h"
#include "vec3.h"

// global constants
#define IMAGE_WIDTH 512
#define IMAGE_HEIGHT 512
#define IMAGE_SIZE (IMAGE_WIDTH*IMAGE_HEIGHT)
#define TILE_SIZE (IMAGE_SIZE)
#define NUM_SAMPLES 1000

#define NUM_SPHERES 8

//#define SAFE_CALL(x) {auto s=std::chrono::steady_clock::now();x;hipDeviceSynchronize(); auto e=std::chrono::stead_clock::now();double diff=std::chrono::duration<double>(e-s).count();if(diff>2.0)printf("-----Possible timeout in %s of %f seconds-----\n",#x,diff);}

struct ray
{
	vec3 o;
	vec3 dir;
};
struct pathState
{
	ray vDir;
	color weight;
	int bounceNum;
	int sampleNum;
};

void checkError()
{
	hipError_t err = hipGetLastError();
	if (hipSuccess != err)
		printf("error! ID: %d, \"%s\"\n", err, hipGetErrorString(err));
}

sceneDesc scene_device;

__device__ ray getCameraRay(int idx)
{
	float y = (int)(idx / IMAGE_WIDTH) - IMAGE_HEIGHT / 2;
	float x = idx % IMAGE_WIDTH - IMAGE_WIDTH / 2;

	ray ret;
	ret.o = vec3(0, 1.f, 5);
	ret.dir = normalized(vec3(x / 2 / IMAGE_WIDTH, y / 2 / IMAGE_HEIGHT, -0.5));
	return ret;
}
__device__ ray randCameraRay(camera* cam, vec3 posRelFilm, hiprandState* crs)
{
	float r = cam->radius * sqrt(hiprand_uniform(crs));
	float theta = 2 * 3.14159 * hiprand_uniform(crs);
	vec3 o = vec3(r * cos(theta), r * sin(theta), 0);

	posRelFilm.z = cam->distFromFilm;
	posRelFilm = posRelFilm * -cam->focalLength / cam->distFromFilm;
	ray ret;
	ret.o = o + cam->pos;
	ret.dir = normalized(posRelFilm - o);

	return ret;
}
__device__ float intersectSphere(vec3 o, vec3 ray, vec3 cpos, float radius)
{
	vec3 nray = normalized(ray);
	float a = dot(ray, ray);
	vec3 oRelCpos = o - cpos;
	float b = 2.0 * dot(ray, oRelCpos);
	float c = dot(oRelCpos, oRelCpos) - radius*radius;
	if (b * b - 4.0 * a * c < 0.0)
		return 0.0;
	float t0 = (-b - sqrt(b * b - 4.0 * a * c)) / (2.0 * a),
		t1 = (-b + sqrt(b * b - 4.0 * a * c)) / (2.0 * a);

	if (t0 <= 0.0 && t1 >= 0.0)
		return t1;
	if (t1 <= 0.0 && t0 >= 0.0)
		return t0;
	return min(t0, t1);
}
__device__ vec3 getTangent(vec3 norm)
{
	vec3 tangent;
	vec3 c1 = cross(norm, vec3(0, 0, 1));
	vec3 c2 = cross(norm, vec3(0, 1, 0));
	if (dot(c1, c1) > dot(c2, c2))
		tangent = c1;
	else
		tangent = c2;
	return tangent;
}

__device__ float nrand(hiprandState* crs)
{
	return hiprand_uniform(crs);
}
__device__ vec3 cosineWeightedRay(vec3 norm, hiprandState* crs) {
	float u1 = nrand(crs),
		u2 = nrand(crs);

	float r_sqr = 1.0 - u1 * u1;
	if (r_sqr < 0.0)
		r_sqr = 0.0;
	float r = sqrt(r_sqr);
	float theta = 2.0 * 3.14159 * u2;

	float x = r * cos(theta);
	float z = r * sin(theta);
	float y = u1;

	vec3 castRay = vec3(x, y, z);
	vec3 tangent = getTangent(norm);
	vec3 bitangent = cross(norm, tangent);
	castRay = norm * castRay.y + 
			tangent * castRay.x + 
			bitangent * castRay.z;
	castRay = normalized(castRay);

	return castRay;
}

__device__ color BRDF(materialDesc m, vec3 vDir, vec3 lDir)
{
	return mul(m.albedo, 1 / 3.14159);
}

__device__ bool radianceAlongSingleStep(pathState* pathState, sceneDesc scene, hiprandState* crs)
{
	if (pathState->bounceNum > 3)
	{
		pathState->weight = color(0, 0, 0);
		return true;
	}

	// intersect
	float closestT = MAX_FLOAT;
	int trisID = -1;
	for (int i = 0; i < scene.numTris; ++i)
	{
		float t = triIntersect(pathState->vDir.o, pathState->vDir.dir, scene.verts, scene.tris, i);
		if (0 < t && t < closestT)
		{
			closestT = t;
			trisID = i;
		}
	}
	closestT -= 0.001;
	if (closestT < 0.001)
	{
		pathState->weight = color(0, 0, 0);
		return true;
	}
	if (closestT > MAX_FLOAT - 1)
	{
		pathState->weight = mul(pathState->weight, color(0, 0, 0));
		return true;
	}

	triangle curTris = scene.tris[trisID];
	materialDesc curMat = scene.mats[curTris.mat];

	pathState->vDir.o = pathState->vDir.o + pathState->vDir.dir * closestT;
	vec3 normal = scene.tris[trisID].norm;

	if (curMat.emmision.r != 0)
	{
		pathState->weight = mul(pathState->weight, curMat.emmision);
		return true;
	}

	vec3 vDirReverse = pathState->vDir.dir * -1;
	vec3 lDir = cosineWeightedRay(normal, crs);

	color curWeight = curMat.albedo;
	pathState->weight = mul(pathState->weight, curWeight);

	pathState->vDir.dir = lDir;
	pathState->bounceNum += 1;
	return false;
}

//-----------------------------------------------------------------------------
// global functions
//-----------------------------------------------------------------------------
__global__ void setupImgBuffer(color* imgBuff)
{
	int idx = blockIdx.x*blockDim.x + threadIdx.x;
	if (idx >= IMAGE_SIZE)
		return;
	imgBuff[idx] = color(0, 0, 0);
}
__global__ void setupPathStateBuffer(pathState* pathStateBuffer, camera* cam, hiprandState* randState)
{
	int idx = blockIdx.x*blockDim.x + threadIdx.x;
	if (idx >= IMAGE_SIZE)
		return;

	float y = (float)((int)(idx / IMAGE_WIDTH)) / IMAGE_HEIGHT - 0.5;
	float x = (float)(idx % IMAGE_WIDTH) / IMAGE_WIDTH - 0.5;

	pathStateBuffer[idx].vDir = randCameraRay(cam, vec3(x, y, 0), &randState[idx]);;
	pathStateBuffer[idx].weight = color(1, 1, 1);
	pathStateBuffer[idx].bounceNum = 0;
	pathStateBuffer[idx].sampleNum = 1;
}
__global__ void setupCurand(hiprandState *state)
{
	int idx = blockIdx.x*blockDim.x + threadIdx.x;
	if (idx >= IMAGE_SIZE)
		return;
	hiprand_init(1234, idx, 0, &state[idx]);
}

__global__ void drawPixel(
	color* imgBuff,
	pathState* pathStateBuffer,
	sceneDesc scene,
	camera* cam,
	hiprandState* randState)
{
	int idx = blockIdx.x*blockDim.x + threadIdx.x;
	if (idx >= IMAGE_SIZE)
		return;

	bool result = radianceAlongSingleStep(&pathStateBuffer[idx], scene, &randState[idx]);
	if (result)
	{
		int curSampleNum = pathStateBuffer[idx].sampleNum;
		color prevSum = denormalized(imgBuff[idx]);
		imgBuff[idx] = add(mul(prevSum, (float)(curSampleNum - 1) / curSampleNum), mul(pathStateBuffer[idx].weight, 1.0f / curSampleNum));

		float y = (float)((int)(idx / IMAGE_WIDTH)) / IMAGE_HEIGHT - 0.5;
		float x = (float)(idx % IMAGE_WIDTH) / IMAGE_WIDTH - 0.5;

		imgBuff[idx] = normalized(imgBuff[idx]);
		pathStateBuffer[idx].vDir = randCameraRay(cam, vec3(x, y, 0), &randState[idx]);
		pathStateBuffer[idx].weight = color(1, 1, 1);
		pathStateBuffer[idx].bounceNum = 0;
		pathStateBuffer[idx].sampleNum += 1;
	}
}

int main()
{
	// load shit
	loadOBJ("models/CornellBox-Original.obj", vec3(), 1);
	//loadOBJ("models/teapot.obj", vec3(0, 1, 0), 1);
	loadOBJ("models/cube.obj", vec3(0, 0, 0), 0.5);
	//loadOBJ("models/my_cornell.obj", vec3(), 1);
	//loadOBJ("models/CornellBox-Sphere.obj", vec3(), 1);
	//buildBVH();

	int nThreads = IMAGE_WIDTH;
	int nblocks = IMAGE_HEIGHT;

	// setup the random number generator
	hiprandState* randState_device;
	hipMalloc((void**)&randState_device, IMAGE_SIZE * sizeof(hiprandState));
	setupCurand <<< nblocks, nThreads >>>(randState_device);

	// set up the camera
	camera cam;
	cam.pos = vec3(0, 1, 3);
	cam.distFromFilm = 1;
	cam.focalLength = 3;
	cam.radius = 0.0;
	camera* cam_device;
	printf("camera size: %zd", sizeof(camera));
	hipMalloc((void**)&cam_device, sizeof(camera));
	hipMemcpy(cam_device, &cam, sizeof(camera), hipMemcpyHostToDevice);

	hipDeviceSynchronize();
	checkError();
	
	// setup host image buffer
	color* imgBuffer_host = (color*)malloc(IMAGE_SIZE * sizeof(color));

	// setup device image buffer
	color* imgBuffer_device;
	hipMalloc((void**)&imgBuffer_device, IMAGE_SIZE * sizeof(color));
	setupImgBuffer <<< nblocks, nThreads >>>(imgBuffer_device);

	// setup path state buffer
	pathState* pathStateBuffer_device;
	hipMalloc((void**)&pathStateBuffer_device, IMAGE_SIZE * sizeof(pathState));
	setupPathStateBuffer <<< nblocks, nThreads >>>(pathStateBuffer_device, cam_device, randState_device);

	// vertex buffer
	hipMalloc((void**)&scene_device.verts, verts.size() * sizeof(vec3));
	hipMemcpy(scene_device.verts, &(verts[0]), verts.size() * sizeof(vec3), hipMemcpyHostToDevice);
	scene_device.numVerts = verts.size();

	// triangle buffer
	hipMalloc((void**)&scene_device.tris, tris.size() * sizeof(triangle));
	hipMemcpy(scene_device.tris, &(tris[0]), tris.size() * sizeof(triangle), hipMemcpyHostToDevice);
	scene_device.numTris = tris.size();

	// material buffer
	hipMalloc((void**)&scene_device.mats, mats.size() * sizeof(materialDesc));
	hipMemcpy(scene_device.mats, &(mats[0]), mats.size() * sizeof(materialDesc), hipMemcpyHostToDevice);
	scene_device.numMats = mats.size();

	hipDeviceSynchronize();
	checkError();

	// render loop
	printf("\nEntering render loop...\n\n");
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start, 0);
	{
		int sampleNum = 1;
		while (sampleNum < NUM_SAMPLES)
		{
			auto start = std::chrono::steady_clock::now();

			drawPixel << < nblocks, nThreads >> >(imgBuffer_device, pathStateBuffer_device, scene_device, cam_device, randState_device);

			if (sampleNum % 10 == 0)
				printf("sample %d finished\n", sampleNum);
			hipDeviceSynchronize();

			auto end = std::chrono::steady_clock::now();
			double diff = std::chrono::duration<double>(end - start).count();
			if (diff > 0.5)
				printf("-----Possible too long execution of %f seconds-----\n", diff);

			checkError();
			++sampleNum;
		}
	}
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop); // shouldnt need this but fuck CUDA
	printf("exiting render loop!\n\n");

	// report time taken
	float milliseconds = 0;
	hipEventElapsedTime(&milliseconds, start, stop);
	printf("Render took %f ms (%f s)\n", milliseconds, milliseconds / 1000);
	printf("%f ms per loop \n", milliseconds / NUM_SAMPLES);

	// build imgBuffer_host
	hipMemcpy(imgBuffer_host, imgBuffer_device, IMAGE_SIZE * sizeof(color), hipMemcpyDeviceToHost);

	// save the file
	FILE* fp = fopen("image.ppm", "w");
	fprintf(fp, "P3 %d %d 255\n", IMAGE_WIDTH, IMAGE_HEIGHT);
	for (int y = 0; y < IMAGE_HEIGHT; ++y)
	{
		for (int x = IMAGE_WIDTH - 1; x >= 0; --x)
		{
			int idx = y * IMAGE_WIDTH + x;
			color c = imgBuffer_host[idx];
			fprintf(fp, "%d %d %d ", (int)(c.r * 255), (int)(c.g * 255), (int)(c.b * 255));
		}
	}
	fclose(fp);

	printf("\nfinished");

	hipFree(randState_device);
	hipFree(cam_device);

	getchar();

	return 0;
}